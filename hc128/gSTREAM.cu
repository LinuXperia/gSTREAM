#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <string.h>
#include <errno.h>
#include "gSTREAM.h"

/* include cipher kernel function cu file */
#include "HC128_kernel.cu"


void gSTREAM_init(gSTREAM_ctx* ctx, int device, int nr_threads, int nr_blocks){

   hipDeviceProp_t deviceProp;
   int nr_streams=nr_threads*nr_blocks;

   /* set device */
   hipGetDeviceProperties(&deviceProp, device);
   hipSetDevice(device);
   debug("\nUsing device %d: \"%s\"\n", device, deviceProp.name);

   cutilSafeCall(hipSetDeviceFlags(hipDeviceMapHost));

   ctx->nr_threads = nr_threads;
   ctx->nr_blocks = nr_blocks;
   ctx->allocated_keys=0;
   ctx->allocated_ivs=0;
   ctx->allocated_buff=0;

   cutilCheckError(cutCreateTimer(&(ctx->bench.timer)));

   /* allocate cipher state */
   HC128_ctx *hctx=&ctx->hctx;
   cutilSafeCall(hipMalloc((void**)&(hctx->P_d),nr_streams*512*sizeof(u32)));
   cutilSafeCall(hipMalloc((void**)&(hctx->Q_d),nr_streams*512*sizeof(u32)));

}

void gSTREAM_exit(gSTREAM_ctx* ctx) {

   if(ctx->allocated_keys) {
      cutilSafeCall(hipFree(ctx->keys_d));
   }

   if(ctx->allocated_ivs) {
      cutilSafeCall(hipFree(ctx->ivs_d));
   }

   if(ctx->allocated_buff) {
      cutilSafeCall(hipHostFree(ctx->buff_h));
   }

   cutilCheckError(cutDeleteTimer(ctx->bench.timer));

   /* free cipher state */
   HC128_ctx *hctx=&ctx->hctx;
   cutilSafeCall(hipFree(hctx->P_d));
   cutilSafeCall(hipFree(hctx->Q_d));
}

void gSTREAM_keysetup(gSTREAM_ctx* ctx, u8* keys, u32 keysize, u32 ivsize) {

   size_t keys_size;
   int nr_streams=ctx->nr_threads*ctx->nr_blocks;
   u32* keys_h=NULL;
   size_t key_size_bytes=sizeof(u8)*(((keysize-1)/(sizeof(u8)*8))+1);
   size_t key_size_nrwords=(((keysize-1)/(sizeof(u32)*8))+1);

   ctx->key_size=keysize;
   ctx->iv_size=ivsize;

   /* allocate keys */
   keys_size=nr_streams*sizeof(u32)*(((keysize-1)/(sizeof(u32)*8))+1);
   cutilSafeCall(hipMalloc((void**)&(ctx->keys_d),keys_size));
   ctx->allocated_keys=1;
   if(!(keys_h=(u32*)malloc(keys_size))) {
      fprintf(stderr,"Could not allocate keys_h: %s\n",strerror(errno));
      exit(-1);
   }

   /* copy byte-aligned keys to word-stream-aligned keys */
   {
      u32  *curr_key;
      u8* tmp_keys=keys;

      /* allocate a current working key */
      if(!(curr_key=(u32*)malloc(sizeof(u32)*key_size_nrwords))) {
         fprintf(stderr,"Could not allocate curr_key: %s\n",strerror(errno));
         exit(-1);
      }
      memset(curr_key,0x00,sizeof(u32)*key_size_nrwords);

      for(int i=0;i<nr_streams;i++) {
         /* copy one of the keys to current key */
         memcpy(curr_key,tmp_keys,key_size_bytes);
         tmp_keys+=key_size_bytes;
         /* copy current key to stream-aligned one */
         for(int j=0;j<key_size_nrwords;j++) {
            keys_h[j*nr_streams+i]=CH_ENDIANESS32(curr_key[j]);
         }
      }

      free(curr_key);
   }


   /* Copy keys to device and free them from host */
   cutilSafeCall(hipMemcpy(ctx->keys_d,keys_h,keys_size,
                                          hipMemcpyHostToDevice));
   free(keys_h);

}

void gSTREAM_ivsetup(gSTREAM_ctx* ctx, u8* ivs) {

   int nr_streams=ctx->nr_threads*ctx->nr_blocks;
   /* initialize the registers to all zeros */

   if(ctx->iv_size>0) {
      u8* tmp_ivs=ivs;
      u32* ivs_h=NULL;
      size_t ivs_size=
         nr_streams*sizeof(u32)*(((ctx->iv_size-1)/(sizeof(u32)*8))+1);

      u32  *curr_iv;
      size_t iv_size_bytes=sizeof(u8)*(((ctx->iv_size-1)/(sizeof(u8)*8))+1);
      size_t iv_size_nrwords=(((ctx->iv_size-1)/(sizeof(u32)*8))+1);

      cutilSafeCall(hipMalloc((void**)&(ctx->ivs_d),ivs_size));
      ctx->allocated_ivs=1;

      if(!(ivs_h=(u32*)malloc(ivs_size))) {
         fprintf(stderr,"Could not allocate ivs_h: %s\n",strerror(errno));
         exit(-1);
      }

      /* allocate a current working iv */
      if(!(curr_iv=(u32*)malloc(sizeof(u32)*iv_size_nrwords))) {
         fprintf(stderr,"Could not allocate curr_iv: %s\n",strerror(errno));
         exit(-1);
      }
      memset(curr_iv,0x00,sizeof(u32)*iv_size_nrwords);

      for(int i=0;i<nr_streams;i++) {
         /* copy one of the ivs to current iv */
         memcpy(curr_iv,tmp_ivs,iv_size_bytes);
         tmp_ivs+=iv_size_bytes;
         /* copy current iv to stream-aligned one */
         for(int j=0;j<iv_size_nrwords;j++) {
            ivs_h[j*nr_streams+i]=CH_ENDIANESS32(curr_iv[j]);
         }
      }
      free(curr_iv);

      /* Copy ivs to device and free them from host */
      cutilSafeCall(hipMemcpy(ctx->ivs_d,ivs_h,ivs_size,
                                                hipMemcpyHostToDevice));
      free(ivs_h);
   }

   /* Load in iv, key and preclock */
   HC128_ctx *hctx=&ctx->hctx;
   size_t smem_size=ctx->nr_threads*17*sizeof(u32);
   HC128_keyivsetup<<<ctx->nr_blocks,ctx->nr_threads,smem_size>>>(hctx->P_d
                                                                 ,hctx->Q_d
                                                                 ,ctx->keys_d
                                                                 ,ctx->key_size
                                                                 ,ctx->ivs_d
                                                                 ,ctx->iv_size);
   cutilCheckMsg("Kernel execution failed");
   hipDeviceSynchronize();

   hctx->counter=0;
#if 0
   {//print state, each colum corresponds to a different stream
      u32 *T_h;
      if(!(T_h=(u32*)malloc(nr_streams*1024*sizeof(u32)))) {
         fprintf(stderr, "Failed to allocate c_h: %s\n",strerror(errno));
         exit(-1);
      }

      cutilSafeCall(hipMemcpy(T_h,hctx->P_d,(nr_streams*512*sizeof(u32)),
               hipMemcpyDeviceToHost));

      cutilSafeCall(hipMemcpy(T_h+512,hctx->Q_d,(nr_streams*512*sizeof(u32)),
               hipMemcpyDeviceToHost));

      int counter=0;
      for(int i=0;i<nr_streams*1024;i++) {
         printf("[%4d : 0x%08x], ",counter, T_h[i]);
         if(!((i+1)%nr_streams)) { printf("\n");counter++; }
      }

      free(T_h);
   }
#endif

}

void gSTREAM_keystream_bytes(gSTREAM_ctx* ctx, u8* keystreams, u32 length) {
   gSTREAM_process_bytes(GEN_KEYSTREAM,ctx,NULL,keystreams,length);
}

void gSTREAM_process_bytes(gSTREAM_action action, gSTREAM_ctx* ctx,
                                       u8* inputs, u8* outputs, u32 length) {
   int nr_streams=ctx->nr_blocks*ctx->nr_threads;
   size_t length_nr_words=(((length-1)/(sizeof(u32)))+1);
   size_t buff_size=nr_streams*length_nr_words*sizeof(u32);
   u32* tmp_buffer;

   /* allocate buffer */
   if((!ctx->allocated_buff)||((length_nr_words*sizeof(u32))>ctx->buff_size)) {
      if(ctx->allocated_buff) {
         free(ctx->buff_h); //alocate a large buffer
      }
      cutilSafeCall(hipHostAlloc((void**)&(ctx->buff_h),buff_size,
               hipHostMallocMapped));
      cutilSafeCall(hipHostGetDevicePointer((void **)&(ctx->buff_d),
               ctx->buff_h,0));
      ctx->allocated_buff=1;
      ctx->buff_size=length_nr_words*sizeof(u32);
   }

   /* allocate a current working buffer */
   if(!(tmp_buffer=(u32*)malloc(sizeof(u32)*length_nr_words))) {
      fprintf(stderr,"Could not allocate tmp_buffer: %s\n",strerror(errno));
      exit(-1);
   }

   if(action!=GEN_KEYSTREAM) {
      for(int i=0;i<nr_streams;i++) {
         /* copy one of the inputs to current working buffer */
         memcpy(tmp_buffer,inputs,length);
         inputs+=length;
         /* copy current iv to stream-aligned one */
         for(int j=0;j<length_nr_words;j++) {
            ctx->buff_h[j*nr_streams+i]=CH_ENDIANESS32(tmp_buffer[j]);
         }
      }
   }

   /* process bytes */
   HC128_ctx *hctx=&ctx->hctx;
   cutilCheckError(cutStartTimer(ctx->bench.timer));
   HC128_process_bytes<<<ctx->nr_blocks,ctx->nr_threads>>>(action
                                                          ,hctx->P_d
                                                          ,hctx->Q_d
                                                          ,ctx->buff_d
                                                          ,hctx->counter
                                                          ,length_nr_words);
   cutilCheckMsg("Kernel execution failed");
   hipDeviceSynchronize();
   cutilCheckError(cutStopTimer(ctx->bench.timer));
   hctx->counter+=length_nr_words;

   /* copy from working buffer to output buffer */
   for(int i=0;i<nr_streams;i++) {
      /* copy one of the keystreams to current keystream */
      for(int j=0;j<length_nr_words;j++) {
         tmp_buffer[j]=ctx->buff_h[i+j*nr_streams];
      }
      memcpy(outputs,tmp_buffer,length);
      outputs+=length;
   }

   free(tmp_buffer);
#if 0
   {//print state, each colum corresponds to a different stream
      u32 *T_h;
      if(!(T_h=(u32*)malloc(nr_streams*1024*sizeof(u32)))) {
         fprintf(stderr, "Failed to allocate c_h: %s\n",strerror(errno));
         exit(-1);
      }

      cutilSafeCall(hipMemcpy(T_h,hctx->P_d,(nr_streams*512*sizeof(u32)),
               hipMemcpyDeviceToHost));
      cutilSafeCall(hipMemcpy(T_h+512,hctx->Q_d,(nr_streams*512*sizeof(u32)),
               hipMemcpyDeviceToHost));

      int counter=0;
      for(int i=0;i<nr_streams*1024;i++) {
         printf("[%4d : 0x%08x], ",counter, T_h[i]);
         if(!((i+1)%nr_streams)) { printf("\n");counter++; }
      }

      free(T_h);
   }
#endif
}

double gSTREAM_getTimerValue(gSTREAM_ctx* ctx) {
   return cutGetTimerValue(ctx->bench.timer);
}
